
#include <hip/hip_runtime.h>
template <int BLOCK_SIZE> __global__ void
matrixMul(float* C, float* A, float* B, int wA, int wB)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = wA * BLOCK_SIZE * by;

    int aEnd   = aBegin + wA - 1;

    int aStep  = BLOCK_SIZE;

    int bBegin = BLOCK_SIZE * bx;

    int bStep  = BLOCK_SIZE * wB;

    float Csub = 0;

    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {

        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        __syncthreads();


        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }
        __syncthreads();
    }

    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

int main(void)
{
    dim3 grid(300, 300);
    dim3 threads(32, 32);
    dim3 dimsA(300*32, 300*32, 1);
    dim3 dimsB(300*32, 300*32, 1);
    float* A, *B, *C;
    
    matrixMul<32><<<grid, threads>>>(C, A, B, 0, 0);
}


#include <hip/hip_runtime.h>
#define TILE_DIM    16
#define BLOCK_ROWS  16

#define FLOOR(a,b) (a-(a%b))

__global__ void transposeNaive(float* odata, float* idata, int width, int height)
{
    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

    int index_in  = xIndex + width * yIndex;
    int index_out = yIndex + height * xIndex;

    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS)
    {
        odata[index_out+i] = idata[index_in+i*width];
    }
}
